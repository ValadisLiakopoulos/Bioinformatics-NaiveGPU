#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h> 
//TODO: Implement dynamic block allocation on the GPU


int string_length(const char *str)
{
    int length = 0;
    while (str[length] != '\0') {
        length++;
    }
    return length;
}

__global__ void naiveSearch(char* sequence, char* pattern, int* seq_len, int* pat_len, int* matches)
{
    int sequence_len = *seq_len;
    int pattern_len = *pat_len;
    int tid = threadIdx.x;
    int chunk_size = (sequence_len )/blockDim.x; //Calculate chunk size for every thread
    int start = tid*chunk_size;
    int end = (tid+1)*chunk_size;
    int local_matches = 0;
    int j; // for iterations

    if (tid == blockDim.x - 1)
        end = sequence_len - pattern_len;


    for(int i = start; i < end; i++) // Iterate through the sequence
    {
        for(j = 0; j < pattern_len; j++)
        {
            if(sequence[i + j] != pattern[j])
                j=pattern_len+10;
        }
        if(j == pattern_len)
        {
            local_matches++; // Increment the local matches
        }
        
    }
    __syncthreads();
    atomicAdd(matches,local_matches); // Add the local matches to the global matches thread-safely
}


int main(int argc, char** argv)
{
    // Open the Sequence and remove any newlines
    FILE *input_file;
    //char ch;
    char* pattern = argv[1];
    int pattern_len = string_length(pattern);
    int file_size = 0;
    int seq_length;
    int thread_num = 1024;
    int matches=0; // Variable to store the integer on the host

    printf("\n\nParallel Version of Naive Pattern Matching Algorithm GPU Accelerated\n");
    // Open the initial sequence for reading

    // input_file = fopen("seq.txt", "r");
    // if (input_file == NULL)
    // {
    //     perror("Error opening the file\n");
    //     return EXIT_FAILURE;
    // }

    // // Open the output file for writing the sequence
    // output_file = fopen("seq_horizontal.txt", "w");
    // if (output_file == NULL)
    // {
    //     perror("Error opening the file\n");
    //     fclose(input_file);
    //     return EXIT_FAILURE;
    // }

    // // Read the sequence from input file and write to output file without newline characters
    // while ((ch = fgetc(input_file)) != EOF)
    // {
    //     if (ch != '\n') {
    //         fputc(ch, output_file);
    //     }
    // }

    // // Close the files
    // rewind(input_file);
    // fclose(input_file);
    // fclose(output_file);
    
    // Open the horizontal sequence for reading
    input_file = fopen("seq_horizontal.txt","r"); 

    //Get the sequence size
    fseek(input_file, 0, SEEK_END);
    file_size = ftell(input_file);
    fseek(input_file, 0, SEEK_SET);

    // Allocate memory for the sequence
    char* sequence = (char *)malloc(file_size*sizeof(char)); 

    if (sequence == NULL) // allocation failed
    {
        perror("Memory allocation for the sequence failed\n");
        fclose(input_file);
        return EXIT_FAILURE;
    }
    else // allocation successful
    {
        fread(sequence, sizeof(char), file_size, input_file);
    }

    seq_length = (int)string_length(sequence);
    fclose(input_file);


    printf("\nSequence Length: %d\nPattern Length: %d\n", seq_length, pattern_len);
    char* gpu_sequence; // Pointer to store the sequence on the device
    char* gpu_pattern; // Pointer to store the pattern on the device
    int *gpu_sequence_length; // Pointer to store the sequence length
    int *gpu_pattern_length; // Pointer to store the pattern length
    int *gpu_matches; // Pointer to store the integer on the device


    hipEvent_t start, stop; // Events for timing the process when executing on the GPU
    hipEventCreate(&start); // Create events for timing the process
    hipEventCreate(&stop);
    hipEventRecord(start); // Start timer to calculate also the time of memory transfer
    hipEventSynchronize(start); // Synchronize the event to start the timer
    // Allocate memory on the GPU
    hipMalloc(&gpu_sequence, seq_length * sizeof(char));
    hipMalloc(&gpu_pattern, pattern_len * sizeof(char));
    hipMalloc(&gpu_matches, sizeof(int));
    hipMalloc(&gpu_sequence_length, sizeof(int));
    hipMalloc(&gpu_pattern_length, sizeof(int));


    
    // Copy the sequence and pattern to the GPU
    hipMemcpy(gpu_sequence, sequence, seq_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(gpu_pattern, pattern, pattern_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(gpu_sequence_length, &seq_length, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_pattern_length, &pattern_len, sizeof(int), hipMemcpyHostToDevice);
    
    // At the moment one block is utilized and 1024 threads - matches the Tesla V100 architecture
    // TODO: Implement dynamic block allocation on the GPU
    naiveSearch<<<1, thread_num>>>(gpu_sequence, gpu_pattern, gpu_sequence_length, gpu_pattern_length, gpu_matches);
    //print_gpu<<<1,1>>>(gpu_matches);
    hipMemcpy(&matches, gpu_matches, sizeof(int), hipMemcpyDeviceToHost);


    hipEventRecord(stop); // End timer to calculate time taken for pattern matching and memory transfer
    hipEventSynchronize(stop);

    float time_ms;

    hipEventElapsedTime(&time_ms, start, stop); // Calculate the time taken

    //TODO: Hold position of matches in an array and print them out

    // Free the memory off the device and host
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(gpu_pattern);
    hipFree(gpu_sequence);
    hipFree(gpu_pattern_length);
    hipFree(gpu_sequence_length);
    hipFree(gpu_matches);

    free(sequence);
    printf("Matches Found: %d\n", matches); // Print the number of matches
    printf("Time taken for pattern matching %f seconds\n\n", time_ms/1000);
    return 0;
}